#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C" {
#endif

#include <stdio.h>
#include <math.h>
#include <float.h>
#include "roi1d_pooling_kernel.h"

#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)

// NCDHW format
__global__ void ROI1DPoolForward(const int nthreads, const float* bottom_data,
    const float temporal_scale, const int num_rois, const int depth,
    const int channels, const int pooled_depth,
    const float* bottom_rois, float* top_data)
{
    CUDA_1D_KERNEL_LOOP(index, nthreads)
    {
        // (n, c, ph, pw) is an element in the pooled output
        int n = index;
        int pd = n % pooled_depth;
        n /= pooled_depth;
        int c = n % channels;
        n /= channels;
        int roi_b = n / num_rois;

        // [start, end) interval for spatial sampling
        bottom_rois += n * 3;
        int roi_batch_ind = bottom_rois[0];
        // int roi_batch_ind = n / num_rois;
        int roi_start_d = round(bottom_rois[1] * temporal_scale);
        int roi_end_d = round(bottom_rois[2] * temporal_scale) - 1;
        if (roi_batch_ind != roi_b and c == 128)
        {
            printf("n=%d, channels=%d, c=%d, num_rois=%d, index=%d, roi_batch_ind=%d, roi_b=%d, roi_start_d=%d, roi_end_d=%d, temporal_scale=%f\n", 
                    n, channels, c, num_rois, index, roi_b, roi_batch_ind, roi_start_d, roi_end_d, temporal_scale);
            return;
        }

        // Force malformed ROIs to be 1x1
        int roi_depth = max(roi_end_d - roi_start_d + 1, 1);
        float bin_size_d = (float)(roi_depth) / (float)(pooled_depth);

        int dstart = (int)(floor((float)(pd) * bin_size_d));
        int dend = (int)(ceil((float)(pd + 1) * bin_size_d));
        float bin_area = max(dend - dstart, 1);

        // Add roi offsets and clip to input boundaries
        dstart = fminf(fmaxf(dstart + roi_start_d, 0), depth);
        dend = fminf(fmaxf(dend + roi_start_d, 0), depth);
        bool is_empty = dend <= dstart || dend <= 0;

        // Define an empty pooling region to be zero
        float out_sum = 0;
        // float maxval = is_empty ? 0 : -FLT_MAX;
        // // If nothing is pooled, argmax = -1 causes nothing to be backprop'd
        // int maxidx = -1;
        bottom_data += (roi_batch_ind * channels + c) * depth;
        for (int d = dstart; d < dend; ++d) {
            out_sum += bottom_data[d];
            // if (bottom_data[bottom_index] > maxval) {
            //     maxval = bottom_data[bottom_index];
            //     maxidx = bottom_index;
            // }
        }
        // top_data[index] = maxval;
        // if (argmax_data != NULL)
        //     argmax_data[index] = maxidx;
        top_data[index] = is_empty? 0. : out_sum/bin_area;
    }
}

int ROI1DPoolForwardLaucher(
    const float* bottom_data, const float temporal_scale,
    const int num_rois, const int depth,
    const int channels, const int pooled_depth,
    const float* bottom_rois, const int batch_size,
    float* top_data, hipStream_t stream)
{
    const int kThreadsPerBlock = 1024;
    const int output_size = batch_size * num_rois * pooled_depth * channels;
    hipError_t err;


    ROI1DPoolForward<<<(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0, stream>>>(
      output_size, bottom_data, temporal_scale, num_rois, depth, channels,
      pooled_depth, bottom_rois, top_data);

    err = hipGetLastError();
    if(hipSuccess != err)
    {
        fprintf( stderr, "cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
        exit( -1 );
    }

    return 1;
}


__global__ void ROI1DPoolBackward(const int nthreads, const float* top_diff,
    const int num_rois, const float temporal_scale,
    const int depth,  const int channels, const int pooled_depth,
    float* bottom_diff, const float* bottom_rois) {
    CUDA_1D_KERNEL_LOOP(index, nthreads)
    {
        // (n, c, ph, pw) is an element in the pooled output
        int n = index;
        int pd = n % pooled_depth;
        n /= pooled_depth;
        int c = n % channels;
        n /= channels;
        int roi_b = n / num_rois;

        // [start, end) interval for spatial sampling
        bottom_rois += n * 3;
        int roi_batch_ind = bottom_rois[0];
        // if (roi_batch_ind != roi_b)
        // {
        //     fprintf("roi_batch_ind is not right !!!\n");
        //     exit( -1 );
        // }

        // int roi_batch_ind = n / num_rois;
        int roi_start_d = round(bottom_rois[1] * temporal_scale);
        int roi_end_d = round(bottom_rois[2] * temporal_scale) - 1;

        // Force malformed ROIs to be 1x1
        int roi_depth = max(roi_end_d - roi_start_d + 1, 1);
        float bin_size_d = (float)(roi_depth) / (float)(pooled_depth);

        int dstart = (int)(floor((float)(pd) * bin_size_d));
        int dend = (int)(ceil((float)(pd + 1) * bin_size_d));
        float bin_area = max(dend - dstart, 1);

        // Add roi offsets and clip to input boundaries
        dstart = fminf(fmaxf(dstart + roi_start_d, 0), depth);
        dend = fminf(fmaxf(dend + roi_start_d, 0), depth);
        bool is_empty = dend <= dstart || dend <= 0;

        // Compute c at bottom
        float* offset_bottom_diff = bottom_diff +
            (roi_batch_ind * channels + c) * depth;
        // float bin_area = (hend - hstart) * (wend - wstart) * (dend - dstart);
        float diff_val = is_empty ? 0. : top_diff[index] / bin_area;
        for (int d = dstart; d < dend; ++d) {
            //caffe_gpu_atomic_add(diff_val, offset_bottom_diff + bottom_index);
            atomicAdd(offset_bottom_diff + d, diff_val);
        }
    }
}

int ROI1DPoolBackwardLaucher(const float* top_diff, const float temporal_scale,
    const int batch_size, const int num_rois, const int depth,
    const int channels, const int pooled_depth,
    const float* bottom_rois, float* bottom_diff, hipStream_t stream)
{
    const int kThreadsPerBlock = 1024;
    const int output_size = batch_size * num_rois * pooled_depth * channels;
    hipError_t err;

    ROI1DPoolBackward<<<(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0, stream>>>(
      output_size, top_diff, num_rois, temporal_scale, depth,
      channels, pooled_depth, bottom_diff, bottom_rois);

    err = hipGetLastError();
    if(hipSuccess != err)
    {
        fprintf( stderr, "cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
        exit( -1 );
    }

    return 1;
}


#ifdef __cplusplus
}
#endif


